#include "hip/hip_runtime.h"
//librerias
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include <iostream>
#include <cstdlib>
#include <pthread.h>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;

//imagen original
Mat src1;
//imagen editada
Mat edit;

// CUDA API error checking macro
static void handleError( hipError_t err, const char *file,  int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line );
        exit( EXIT_FAILURE );
    }
}
#define cudaCheck( err ) (handleError( err, __FILE__, __LINE__ ))

__global__ void stencil_1d(int *in , int *out, int di, int repetir){
  //index del hilo actual
  int gindex = repetir +threadIdx.x + (blockIdx.x * blockDim.x);
  int aux = 0;

  //suma de todos los valores desde di a la izquierda hasta di a la derecha del pixel en cuestion
  for (int offset = -di ; offset <= di ; offset++){
    if(gindex+offset>0)
      aux += in[gindex+offset];
    else
      aux += in[gindex-offset];
  }


  //promedio de pixeles sumados
  aux = int(aux/(di*2+1));

  //guardado del valor a enviar al host del nuevo valor del pixel
  out[gindex] = aux;


}

int main ( int argc, char** argv ) {

  //lectura de la imagen que será procesada
  src1 = imread(argv[1], CV_LOAD_IMAGE_UNCHANGED);
  edit = imread(argv[1], CV_LOAD_IMAGE_UNCHANGED);
  //lectura del tamaño del kernel a realizar
  int distorcion = atoi(argv[2]);
  //lectura de la cantidad de cores que serán lanzados
  int NUM_THREADS = atoi(argv[4]);

  //lectura de la cantidad de cores que serán lanzados
  int NUM_BLOCKS = atoi(argv[3]);

  //variables de host
  int k,j,i, h_red_in[src1.cols],h_red_out[src1.cols],repetir;
  //variables de device
  int *d_red_in, *d_red_out;


  //Reserva de recursos en device
  hipMalloc( (void **) &d_red_in, src1.cols * sizeof(int));
  hipMalloc( (void **) &d_red_out, src1.cols * sizeof(int));


  //k recorre fila por fila
  for(k=0;k<src1.rows;k++){
    //i recorre los valores de rgb e.d 0=r 1=g 2=b
    for(i=0;i<=2;i++){
      //asigna los valores de la fila actual en el host
      for(j=0;j<src1.cols;j++)
        h_red_in[j] = int(src1.at<Vec3b>(k,j)[i]);

      //envia los valores de la fila actual del host al device
      cudaCheck( hipMemcpy( d_red_in, h_red_in, src1.cols * sizeof(int), hipMemcpyHostToDevice));

      for(repetir=0;repetir<=src1.cols/(NUM_BLOCKS*NUM_THREADS);repetir++)
        stencil_1d<<<NUM_BLOCKS,NUM_THREADS>>> (d_red_in,d_red_out,distorcion,repetir*(NUM_BLOCKS*NUM_THREADS));

      //guarda en el host los valores generados por el stencil
      hipMemcpy( h_red_out, d_red_out, src1.cols * sizeof(int), hipMemcpyDeviceToHost);

      //recorre la fila actual y le asigna los nuevos valores rgb
      for(j=0;j<src1.cols;j++)
        edit.at<Vec3b>(k,j)[i] = h_red_out[j];

    }
  }

  //k recorre fila por columna
  for(k=0;k<src1.cols;k++){
    //i recorre los valores de rgb e.d 0=r 1=g 2=b
    for(i=0;i<=2;i++){
      //asigna los valores de la columna actual en el host
      for(j=0;j<src1.rows;j++)
        h_red_in[j] = int(src1.at<Vec3b>(j,k)[i]);

      //envia los valores de la columna actual del host al device
      cudaCheck( hipMemcpy( d_red_in, h_red_in, src1.rows * sizeof(int), hipMemcpyHostToDevice));

      for(repetir=0;repetir<=src1.rows/(NUM_BLOCKS*NUM_THREADS);repetir++)
        stencil_1d<<<NUM_BLOCKS,NUM_THREADS>>> (d_red_in,d_red_out,distorcion,repetir*(NUM_BLOCKS*NUM_THREADS));

      //guarda en el host los valores generados por el stencil
      hipMemcpy( h_red_out, d_red_out, src1.rows * sizeof(int), hipMemcpyDeviceToHost);

      //recorre la columna actual y le asigna los nuevos valores rgb a partir de el valor obtenido con el valor previo
      for(j=0;j<src1.rows;j++)
        edit.at<Vec3b>(j,k)[i] = (int(edit.at<Vec3b>(j,k)[i])+h_red_out[j])/2;

    }
  }

  //libera la memoria del device
  hipFree(d_red_out);
  hipFree(d_red_in);

  //genera la imagen
  imwrite( "edicion.jpg", edit );

  return 0;
}
